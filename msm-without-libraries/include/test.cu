#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " \
                      << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

int main() {
    size_t count = 524296;

    // Allocate host-pinned memory
    void* host_ptr;
    CUDA_CHECK(hipHostMalloc(&host_ptr, count, hipHostMallocDefault));

    // Allocate device memory
    void* device_ptr;
    CUDA_CHECK(hipMalloc(&device_ptr, count));

    // Fill device memory with something (e.g. 0s)
    CUDA_CHECK(hipMemset(device_ptr, 0, count));

    // Create non-blocking stream
    hipStream_t streamT;
    CUDA_CHECK(hipStreamCreateWithFlags(&streamT, hipStreamNonBlocking));

    // Perform async copy
    CUDA_CHECK(hipMemcpyAsync(host_ptr, device_ptr, count, hipMemcpyDeviceToHost, streamT));

    // Wait for stream
    CUDA_CHECK(hipStreamSynchronize(streamT));

    // Clean up
    CUDA_CHECK(hipHostFree(host_ptr));
    CUDA_CHECK(hipFree(device_ptr));
    CUDA_CHECK(hipStreamDestroy(streamT));

    std::cout << "Success!" << std::endl;
    return 0;
}
