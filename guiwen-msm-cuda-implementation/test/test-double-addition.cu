#include "hip/hip_runtime.h"
struct instance_params;
struct h_instance_params;

#include <pthread.h>

typedef pthread_t CUTThread;
typedef void *(*CUT_THREADROUTINE)(void *);

#define CUT_THREADPROC void
#define  CUT_THREADEND

//Create thread
CUTThread start_thread(CUT_THREADROUTINE func, void * data){
    pthread_t thread;
    pthread_create(&thread, NULL, func, data);
    return thread;
}

//Wait for thread to finish
void end_thread(CUTThread thread){
    pthread_join(thread, NULL);
}

//Destroy thread
void destroy_thread( CUTThread thread ){
    pthread_cancel(thread);
}

//Wait for multiple threads
void wait_for_threads(const CUTThread * threads, int num){
    for(int i = 0; i < num; i++)
        end_thread( threads[i] );
}



#include <stdio.h>
#include <math.h>
#include <string.h>

#include "../depends/libstl-cuda/memory.cuh"
#include "../depends/libstl-cuda/vector.cuh"
#include "../depends/libstl-cuda/utility.cuh"

#include "../depends/libff-cuda/fields/bigint_host.cuh"
#include "../depends/libff-cuda/fields/fp_host.cuh"
#include "../depends/libff-cuda/fields/fp2_host.cuh"
#include "../depends/libff-cuda/fields/fp6_3over2_host.cuh"
#include "../depends/libff-cuda/fields/fp12_2over3over2_host.cuh"
#include "../depends/libff-cuda/curves/bls12_381/bls12_381_init_host.cuh"
#include "../depends/libff-cuda/curves/bls12_381/bls12_381_g1_host.cuh"
#include "../depends/libff-cuda/curves/bls12_381/bls12_381_g2_host.cuh"
#include "../depends/libff-cuda/curves/bls12_381/bls12_381_pp_host.cuh"
#include "../depends/libmatrix-cuda/transpose/transpose_ell2csr.cuh"
#include "../depends/libmatrix-cuda/spmv/csr-balanced.cuh"
#include "../depends/libff-cuda/scalar_multiplication/multiexp.cuh"


#include "../depends/libff-cuda/curves/bls12_381/bls12_381_init.cuh"
#include "../depends/libff-cuda/curves/bls12_381/bls12_381_pp.cuh"

#include <time.h>

using namespace libff;



#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(const char *const file, int const line)
{
    hipError_t const err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        
        //std::exit(EXIT_FAILURE);
    }
}

struct instance_params
{
    bls12_381_Fr instance;
    bls12_381_G1 g1_instance;
    bls12_381_G2 g2_instance;
    bls12_381_GT gt_instance;
};

struct h_instance_params
{
    bls12_381_Fr_host h_instance;
    bls12_381_G1_host h_g1_instance;
    bls12_381_G2_host h_g2_instance;
    bls12_381_GT_host h_gt_instance;
};


template<typename ppT>
struct MSM_params
{
    libstl::vector<libff::Fr<ppT>> vf;
    libstl::vector<libff::G1<ppT>> vg;
};


__global__ void init_params()
{
    gmp_init_allocator_();
    bls12_381_pp::init_public_params();
}

__global__ void instance_init(instance_params* ip)
{
    ip->instance = bls12_381_Fr(&bls12_381_fp_params_r);
    ip->g1_instance = bls12_381_G1(&g1_params);
    ip->g2_instance = bls12_381_G2(&g2_params);
    ip->gt_instance = bls12_381_GT(&bls12_381_fp12_params_q);
}

void instance_init_host(h_instance_params* ip)
{
    ip->h_instance = bls12_381_Fr_host(&bls12_381_fp_params_r_host);
    ip->h_g1_instance = bls12_381_G1_host(&g1_params_host);
    ip->h_g2_instance = bls12_381_G2_host(&g2_params_host);
    ip->h_gt_instance = bls12_381_GT_host(&bls12_381_fp12_params_q_host);
}


template<typename ppT>
__global__ void generate_MP(MSM_params<ppT>* mp, instance_params* ip, size_t size)
{
    new ((void*)mp) MSM_params<ppT>();
    mp->vf.presize(size, 512, 32);
    mp->vg.presize(size, 512, 32);

    libstl::launch<<<512, 32>>>
    (
        [=]
        __device__ ()
        {
            size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
            size_t tnum = blockDim.x * gridDim.x;
            libff::Fr<ppT> f = ip->instance.random_element();
            libff::G1<ppT> g = ip->g1_instance.random_element();
            f ^= idx;
            g = g * idx;
            while(idx < size)
            {
                mp->vf[idx] = f;
                mp->vg[idx] = g;
                f = f + f;
                g = g + g;
                idx += tnum;
            }
        }
    );
    hipDeviceSynchronize();

    ip->g1_instance.p_batch_to_special(mp->vg, 160, 32);
}

struct Mem
{
    size_t device_id;
    void* mem;
};

void* multi_init_params(void* params)
{
    Mem* device_mem = (Mem*) params;
    hipSetDevice(device_mem->device_id);
    size_t init_size = 1024 * 1024 * 1024;
    init_size *= 15;
    CHECK_LAST_CUDA_ERROR();
    if( hipMalloc( (void**)&device_mem->mem, init_size ) != hipSuccess) printf("device malloc error!\n");
    libstl::initAllocator(device_mem->mem, init_size);
    init_params<<<1, 1>>>();
    hipDeviceSynchronize();
    CHECK_LAST_CUDA_ERROR();
    return 0;
}

struct Instance
{
    size_t device_id;
    instance_params** ip;
};

void* multi_instance_init(void* instance)
{
    Instance* it = (Instance*)instance;
    hipSetDevice(it->device_id);
    CHECK_LAST_CUDA_ERROR();
    if( hipMalloc( (void**)it->ip, sizeof(instance_params)) != hipSuccess) printf("ip malloc error!\n");
    CHECK_LAST_CUDA_ERROR();
    instance_init<<<1, 1>>>(*it->ip);
    hipDeviceSynchronize();
    return 0;
}

template<typename ppT>
struct MSM
{
    size_t device_id;
    MSM_params<ppT>* mp;
    instance_params* ip;
    libff::G1<ppT>* res;
};

template<typename ppT>
void* multi_MSM(void* msm)
{
    MSM<ppT>* it = (MSM<ppT>*)msm;
    hipSetDevice(it->device_id);

    size_t lockMem;
    libstl::lock_host(lockMem);
    libff::p_multi_exp_faster_multi_GPU_host<libff::G1<ppT>, libff::Fr<ppT>, libff::multi_exp_method_naive_plain>(it->mp->vg, it->mp->vf, it->ip->instance, it->ip->g1_instance, 512, 32);
    hipDeviceSynchronize();
    libff::p_multi_exp_faster_multi_GPU_host<libff::G1<ppT>, libff::Fr<ppT>, libff::multi_exp_method_naive_plain>(it->mp->vg, it->mp->vf, it->ip->instance, it->ip->g1_instance, 512, 32);
    hipDeviceSynchronize();
    libff::p_multi_exp_faster_multi_GPU_host<libff::G1<ppT>, libff::Fr<ppT>, libff::multi_exp_method_naive_plain>(it->mp->vg, it->mp->vf, it->ip->instance, it->ip->g1_instance, 512, 32);
    hipDeviceSynchronize();
    libstl::resetlock_host(lockMem);

    hipEvent_t eventMSMStart, eventMSMEnd;
    hipEventCreate( &eventMSMStart);
	hipEventCreate( &eventMSMEnd);
    hipEventRecord( eventMSMStart, 0); 
    hipEventSynchronize(eventMSMStart);
    for(size_t i=0; i<1; i++)
    {
        it->res = libff::p_multi_exp_faster_multi_GPU_host<libff::G1<ppT>, libff::Fr<ppT>, libff::multi_exp_method_naive_plain>(it->mp->vg, it->mp->vf, it->ip->instance, it->ip->g1_instance, 512, 32);
        hipDeviceSynchronize();
    }


    hipEventRecord( eventMSMEnd, 0);
    hipEventSynchronize(eventMSMEnd);
    float   TimeMSM;
    hipEventElapsedTime( &TimeMSM, eventMSMStart, eventMSMEnd );
    printf( "Time thread %lu for MSM:  %3.5f ms\n", it->device_id, TimeMSM );

    return 0;
}

template<typename ppT_host, typename ppT_device>
void D2H(libff::G1<ppT_host>* hg1, libff::G1<ppT_device>* dg1, libff::G1<ppT_host>* g1_instance)
{
    hipMemcpy(hg1, dg1, sizeof(libff::G1<ppT_device>), hipMemcpyDeviceToHost);
    hg1->set_params(g1_instance->params);
}


template<typename ppT>
void Reduce(libff::G1<ppT>* hg1, libff::Fr<ppT>* instance, size_t total)
{
    int device_count;
    hipGetDeviceCount(&device_count);
    
    libff::G1<ppT> g1 = hg1[device_count-1];

    if(device_count != 1)
    {
        for(size_t i=device_count - 2; i <= device_count - 1; i--)
        {
            size_t log2_total = libff::log2(total);
            size_t c = log2_total - (log2_total / 3 - 2);
            size_t num_bits = instance->size_in_bits();
            size_t num_groups = (num_bits + c - 1) / c;
            size_t sgroup = (num_groups + device_count - 1) / device_count * i;
            size_t egroup = (num_groups + device_count - 1) / device_count * (i + 1);
            if(egroup > num_groups) egroup = num_groups;
            if(sgroup > num_groups) sgroup = num_groups;
            if(egroup == sgroup) continue;

            for(size_t j=0; j < (egroup - sgroup) * c; j++)
            {
                g1 = g1.dbl();
            }
            g1 = g1 + hg1[i];
        }
    }

    g1.to_special();

}

__global__ void test_multiplication(instance_params* ip)
{
    bigint<4> a("40717440697516928871729794459926083509479925425253447106356521628001304033345");
    bigint<4> b("40717440697516928871729794459926083509479925425253447106356521628001304033345");
    libff::Fr<bls12_381_pp> f = ip->instance.random_element();
    libff::Fr<bls12_381_pp> g = ip->instance.random_element();
    
    printf("F\n");
    f.as_bigint().print();
    printf("G=\n");
    g.as_bigint().print();
    printf("F*G = \n");
    f = f*g;
    f.as_bigint().print();

    printf("Modulus:\n");
    ip->instance.params->modulus->print();
    
    
    //libff::G1<bls12_381_pp> q = ip->g1_instance.random_element();
    
    libff::bigint<6L> x_val("3455274221085403918568630834754176304780762356539104640283219223328938319396495915575948888146901796993445978399141");
    libff::bigint<6L> y_val("3112993911598655673602636130584638258648881928712239965254047157896709175076195307280186219428719718278126484846253");
    libff::bigint<6L> z_val("1");
    
    libff::bls12_381_Fq m(ip->g1_instance.params->fq_params, x_val);
    libff::bls12_381_Fq n(ip->g1_instance.params->fq_params, y_val);
    libff::bls12_381_Fq o(ip->g1_instance.params->fq_params, z_val);

    libff::G1<bls12_381_pp> generator(ip->g1_instance.params, m,n,o);


    

    // q.to_affine_coordinates();
    printf("X : \n");
    generator.X.as_bigint().print();
    printf("\nY : \n");
    generator.Y.as_bigint().print();
    printf("\nZ : \n");
    generator.Z.as_bigint().print();
    generator = generator.dbl();
    generator.to_affine_coordinates();
    printf("X : \n");
    generator.X.as_bigint().print();
    printf("\nY : \n");
    generator.Y.as_bigint().print();
    printf("\nZ : \n");
    generator.Z.as_bigint().print();
    
}

int main(int argc, char* argv[])
{
    if (argc < 2) {
		printf("Please enter the MSM scales (e.g. 20 represents 2^20) \n");
		return 1;
	}

    int log_size = atoi(argv[1]);

    int deviceCount;
    hipGetDeviceCount( &deviceCount );
    CUTThread  thread[deviceCount];

    bls12_381_pp_host::init_public_params();
    hipSetDevice(0);
    hipFree(0);
    size_t num_v = (size_t) (1 << log_size);

    // params init 
    Mem device_mem[deviceCount];

    printf("%s %d\n",__FILE__, __LINE__);
    CHECK_LAST_CUDA_ERROR();
    for(size_t i=0; i<deviceCount; i++)
    {
        device_mem[i].device_id = i;

        CHECK_LAST_CUDA_ERROR();
        device_mem[i].mem = NULL;

        CHECK_LAST_CUDA_ERROR();
        thread[i] = start_thread( multi_init_params, &device_mem[i] );

        CHECK_LAST_CUDA_ERROR();
    }

    printf("%s %d\n",__FILE__, __LINE__);
    for(size_t i=0; i<deviceCount; i++)
    {
        end_thread(thread[i]);
    }

    // printf("Device count: %d\n", deviceCount);
    // instance init
    instance_params* ip[deviceCount];

    CHECK_LAST_CUDA_ERROR();
    Instance instance[deviceCount];

    printf("%s %d\n",__FILE__, __LINE__);
    CHECK_LAST_CUDA_ERROR();
    for(size_t i=0; i<deviceCount; i++)
    {
        instance[i].device_id = i;

        CHECK_LAST_CUDA_ERROR();
        instance[i].ip = &ip[i];

        CHECK_LAST_CUDA_ERROR();
        thread[i] = start_thread( multi_instance_init, &instance[i] );
    }
    printf("%s %d\n",__FILE__, __LINE__);
    CHECK_LAST_CUDA_ERROR();
    for(size_t i=0; i<deviceCount; i++)
    {
        end_thread(thread[i]);
    }

    printf("%s %d\n",__FILE__, __LINE__);
    CHECK_LAST_CUDA_ERROR();
    h_instance_params hip;

    CHECK_LAST_CUDA_ERROR();
    instance_init_host(&hip);


    printf("%s %d\n",__FILE__, __LINE__);

    test_multiplication<<<1,1>>>(ip[0]);

    // CHECK_LAST_CUDA_ERROR();
    // // elements generation
    // MSM_params<bls12_381_pp>* mp[deviceCount];

    // printf("%s %d\n",__FILE__, __LINE__);
    // CHECK_LAST_CUDA_ERROR();
    // for(size_t i=0; i<deviceCount; i++)
    // {
    //     hipSetDevice(i);
    //     CHECK_LAST_CUDA_ERROR();
    //     hipDeviceSynchronize();
    //     CHECK_LAST_CUDA_ERROR();
    //     if( hipMalloc( (void**)&mp[i], sizeof(MSM_params<bls12_381_pp>)) != hipSuccess) printf("mp malloc error!\n");
    //     CHECK_LAST_CUDA_ERROR();
    //     generate_MP<bls12_381_pp><<<1, 1>>>(mp[i], ip[i], num_v);

    //     CHECK_LAST_CUDA_ERROR();
    // }

    // printf("%s %d\n",__FILE__, __LINE__);
    // for(size_t i=0; i<deviceCount; i++)
    // {
    //     hipSetDevice(i);
    //     hipDeviceSynchronize();
    // }
    // hipSetDevice(0);
    
    // // msm
    // MSM<bls12_381_pp> msm[deviceCount];
    // for(size_t i=0; i<deviceCount; i++)
    // {
    //     msm[i].device_id = i;
    //     msm[i].mp = mp[i];
    //     msm[i].ip = ip[i];
    //     thread[i] = start_thread( multi_MSM<bls12_381_pp>, &msm[i] );
    // }

    // printf("%s %d\n",__FILE__, __LINE__);
    // for(size_t i=0; i<deviceCount; i++)
    // {
    //     end_thread(thread[i]);
    // }

    // printf("%s %d\n",__FILE__, __LINE__);
    // libff::G1<bls12_381_pp_host> hg1[deviceCount];
    // for(size_t i=0; i < deviceCount; i++)
    // {
    //     hipSetDevice(i);
    //     D2H<bls12_381_pp_host, bls12_381_pp>(&hg1[i], msm[i].res, &hip.h_g1_instance);
    // }

    // Reduce<bls12_381_pp_host>(hg1, &hip.h_instance, num_v);

    hipDeviceReset();
    return 0;
}
