#include "hip/hip_runtime.h"
#ifndef __MSM_CUH
#define __MSM_CUH

#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>

#include <hipcub/hipcub.hpp>

#include "./../include/Point.cuh"

#define debug 1

#define WINDOW_SIZE 16
#define NUM_BITS 256
#define CUDA_CHECK(call)                                                         \
    do                                                                           \
    {                                                                            \
        hipError_t err = call;                                                  \
        if (err != hipSuccess)                                                  \
        {                                                                        \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " \
                      << hipGetErrorString(err) << std::endl;                   \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    } while (0)

__global__ void process_scalar_into_bucket(Scalar *scalar,
                                           Point *points,
                                           size_t num_points,
                                           size_t num_window,
                                           uint32_t *scalar_chunks,
                                           uint32_t *indices,
                                           uint32_t *offset,
                                           uint32_t *count)
{
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    while (idx < num_points)
    {

        for (int current_window = 0; current_window < num_window; current_window++)
        {
            size_t bindex = 0;
            size_t start = current_window * WINDOW_SIZE;
            size_t end = start + WINDOW_SIZE;
            for (size_t i = start, j = 0; i < end; i++, j++)
            {
                if (scalar[idx].test_bit(i))
                {
                    bindex |= (1 << j);
                }
            }
            scalar_chunks[idx + current_window * num_points] = bindex;
            if (bindex != 0)
                atomicAdd(&count[bindex + current_window * ((size_t)1 << WINDOW_SIZE)], 1);
        }

        idx += stride;
    }
}

__global__ void construct_bucket_indices(
    const __restrict__ uint32_t *scalar_chunks,
    uint32_t *indices,
    uint32_t *offset_counter,
    size_t num_points,
    size_t num_bucket)
{
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    size_t curr_window = blockIdx.y;
    while (idx < num_points)
    {
        uint32_t bindex = scalar_chunks[idx + curr_window * num_points];
        if (bindex != 0)
            indices[atomicAdd(&offset_counter[bindex + curr_window * num_bucket], 1)] = idx;
        idx += stride;
    }
}

// Sum the bucket points. Needs balancing here to improve performance.
__global__ void sum_buckets(Point *point, Point *sum, uint32_t *offset, uint32_t *indices, uint32_t *count, size_t num_bucket)
{
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t stride = blockDim.x * gridDim.x;
    size_t curr_window = blockIdx.y;
    while (idx < num_bucket)
    {
        Point lsum;
        lsum = lsum.zero();
        // Skip the zero bucket
        if (idx == 0)
        {
            sum[idx + curr_window * num_bucket] = lsum;
            idx += stride;
            continue;
        }
        // If the bucket is empty, skip it
        if (count[idx + curr_window * num_bucket] == 0)
        {
            sum[idx + curr_window * num_bucket] = lsum;
            idx += stride;
            continue;
        }
        for (size_t i = offset[idx + curr_window * num_bucket]; i < offset[idx + curr_window * num_bucket] + count[idx + curr_window * num_bucket]; i++)
        {
            lsum = lsum.mixed_add(point[indices[i]]);
        }
        // lsum = lsum * idx;

        sum[idx + curr_window * num_bucket] = lsum;

        idx += stride;
    }
}
// sum all the buckets of a window and store the result to a point
__global__ void gather_bucket_parallel(Point *sum, size_t bcount, Point *res)
{
    int idx = threadIdx.x;
    int curr_window = blockIdx.x;
    extern __shared__ Point local_sum[];
    Point running_sum;
    Point result;
    int per_thread = (bcount + blockDim.x - 1) / blockDim.x;
    int start = idx * per_thread + 1;
    int end = start + per_thread;

    if (end > bcount)
        end = bcount;
    local_sum[idx] = local_sum[idx].zero();
    running_sum = running_sum.zero();
    result = result.zero();

    for (int i = end - 1; i >= start; i--)
    {
        running_sum = running_sum + sum[i + curr_window * bcount];
        local_sum[idx] = local_sum[idx] + running_sum;
    }

    __syncthreads();

    local_sum[idx] = local_sum[idx] + running_sum * (per_thread * idx);
    __syncthreads();
    if (idx % 2 == 0)
    {
        local_sum[idx] = local_sum[idx] + local_sum[idx + 1];
    }
    __syncthreads();
    if (idx % 4 == 0)
    {
        local_sum[idx] = local_sum[idx] + local_sum[idx + 2];
    }
    __syncthreads();
    if (idx % 8 == 0)
    {
        local_sum[idx] = local_sum[idx] + local_sum[idx + 4];
    }
    __syncthreads();
    if (idx % 16 == 0)
    {
        local_sum[idx] = local_sum[idx] + local_sum[idx + 8];
    }
    __syncthreads();
    if (idx % 32 == 0)
    {
        local_sum[idx] = local_sum[idx] + local_sum[idx + 16];
    }
    __syncthreads();
    if (idx == 0)
    {
        for (int i = 0; i < blockDim.x; i += 32)
        {
            result = result + local_sum[i];
        }
        res[curr_window] = result;
    }
}
// accumulate all window output
__global__ void accumulate_result(Point *window_res, size_t num_window, Point *res)
{
    Point acc = acc.zero();
    for (int i = num_window - 1; i >= 0; i--)
    {
        for (int j = 0; j < WINDOW_SIZE; j++)
        {
            acc = acc.dbl();
        }
        acc = acc + window_res[i];
    }
    *res = acc;
}

#if debug
__global__ void print_point(Point *p, size_t num = 1)
{
    for (size_t i = 0; i < num; i++)
    {
        if (!p[i].is_zero())
        {
            p[i].to_affine();
            p[i].print();
        }
    }
}

#endif

// driver function to perform multi scalar multiplication
void cuda_pippenger_msm(Point *points, Scalar *scalars, size_t num_points)
{
    int num_windows = (NUM_BITS + WINDOW_SIZE - 1) / WINDOW_SIZE;
    size_t num_bucket = ((size_t)1 << WINDOW_SIZE);

    uint32_t *scalar_chunks, *indices; // scalar_chunks put all the scalar chunks in a single array
    // indices array to store the indices of scalars according to bucket
    uint32_t *offset, *offset_counter; // offset for bucket
    uint32_t *count;                   // count for bucket
    uint32_t *h_count;

    CUDA_CHECK(hipMalloc(&scalar_chunks, sizeof(uint32_t) * num_points * num_windows));
    CUDA_CHECK(hipMalloc(&indices, sizeof(uint32_t) * num_points * num_windows));
    CUDA_CHECK(hipMalloc(&offset, sizeof(uint32_t) * num_bucket * num_windows));
    CUDA_CHECK(hipMalloc(&offset_counter, sizeof(uint32_t) * num_bucket * num_windows));
    CUDA_CHECK(hipMalloc(&count, sizeof(uint32_t) * num_bucket * num_windows));
    CUDA_CHECK(hipMemset(count, 0, sizeof(uint32_t) * num_bucket * num_windows));
    CUDA_CHECK(hipMemset(offset, 0, sizeof(uint32_t) * num_bucket * num_windows));
    CUDA_CHECK(hipMemset(offset_counter, 0, sizeof(uint32_t) * num_bucket * num_windows));
    CUDA_CHECK(hipMemset(scalar_chunks, 0, sizeof(uint32_t) * num_points * num_windows));
    CUDA_CHECK(hipMemset(indices, 0, sizeof(uint32_t) * num_points * num_windows));

    h_count = new uint32_t[num_bucket * num_windows];

    size_t blockSize = 256;
    size_t gridSize = (num_points + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    hipEventSynchronize(start);

    // Scan the scalars and construct bucket element counts
    process_scalar_into_bucket<<<gridSize, blockSize>>>(scalars, points, num_points, num_windows, scalar_chunks, indices, offset, count);
    CUDA_CHECK(hipDeviceSynchronize());

    // perform exclusive scan on the count array to get the offsets
    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;

    hipcub::DeviceScan::ExclusiveSum(
        d_temp_storage, temp_storage_bytes, count, offset, num_bucket * num_windows);
    CUDA_CHECK(hipMalloc(&d_temp_storage, temp_storage_bytes));
    hipcub::DeviceScan::ExclusiveSum(
        d_temp_storage, temp_storage_bytes, count, offset, num_bucket * num_windows);
    CUDA_CHECK(hipFree(d_temp_storage));

    // Build indices for each bucket
    CUDA_CHECK(hipMemcpy(offset_counter, offset, sizeof(uint32_t) * num_bucket * num_windows, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipDeviceSynchronize());
    dim3 grid_size(gridSize, num_windows);
    construct_bucket_indices<<<grid_size, blockSize, 0, 0>>>(scalar_chunks, indices, offset_counter, num_points, num_bucket);
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(h_count, count, sizeof(uint32_t) * num_bucket * num_windows, hipMemcpyDeviceToHost));

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    printf("Time taken to process scalar: %f\n", ms);

    // Sum the buckets
    Point *sum;
    CUDA_CHECK(hipMalloc(&sum, sizeof(Point) * num_bucket * num_windows));
    dim3 block(32);
    dim3 grid(512, num_windows); //(num_bucket + blockSize - 1) / blockSize
    sum_buckets<<<grid, block>>>(points, sum, offset, indices, count, num_bucket);
    CUDA_CHECK(hipDeviceSynchronize());

    hipEvent_t sum_stop;
    hipEventCreate(&sum_stop);
    hipEventRecord(sum_stop);
    hipEventSynchronize(sum_stop);
    float sum_ms;
    hipEventElapsedTime(&sum_ms, stop, sum_stop);
    printf("Time taken to sum buckets: %f\n", sum_ms);

    Point *window_res;
    CUDA_CHECK(hipMalloc(&window_res, sizeof(Point) * num_windows));
    dim3 gather_grid(num_windows, 1);
    CUDA_CHECK(hipDeviceSynchronize());
    gather_bucket_parallel<<<gather_grid, 256, 48 * 1024>>>(sum, num_bucket, window_res);
    CUDA_CHECK(hipDeviceSynchronize());

    hipEvent_t gather_stop;
    hipEventCreate(&gather_stop);
    hipEventRecord(gather_stop);
    hipEventSynchronize(gather_stop);
    float gather_ms;
    hipEventElapsedTime(&gather_ms, sum_stop, gather_stop);
    printf("Time taken to gather buckets: %f\n", gather_ms);

    Point *res;
    CUDA_CHECK(hipMalloc(&res, sizeof(Point)));
    accumulate_result<<<1, 1>>>(window_res, num_windows, res);
    CUDA_CHECK(hipDeviceSynchronize());

    hipEvent_t end;
    hipEventCreate(&end);
    hipEventRecord(end);
    hipEventSynchronize(end);

    float accumulate_ms;
    hipEventElapsedTime(&accumulate_ms, gather_stop, end);
    printf("Time taken to accumulate result: %f\n", accumulate_ms);

    float total_time;
    hipEventElapsedTime(&total_time, start, end);
    printf("Total time taken: %f ms\n", total_time);

#if debug
    print_point<<<1, 1>>>(res, 1);
    CUDA_CHECK(hipDeviceSynchronize());
#endif

    // Free memory
    CUDA_CHECK(hipFree(scalar_chunks));
    CUDA_CHECK(hipFree(indices));
    CUDA_CHECK(hipFree(offset));
    CUDA_CHECK(hipFree(offset_counter));
    CUDA_CHECK(hipFree(count));
    CUDA_CHECK(hipFree(sum));
    CUDA_CHECK(hipFree(window_res));

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(end);
}

#endif