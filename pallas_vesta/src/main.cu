#include "hip/hip_runtime.h"
#ifndef __MAIN_RUN
#define __MAIN_RUN

#include <stdio.h>

#include "./../include/Field.cuh"
#include "./../include/Point.cuh"



#include <pthread.h>

typedef pthread_t CUTThread;
typedef void *(*CUT_THREADROUTINE)(void *);

#define CUT_THREADPROC void
#define  CUT_THREADEND

//Create thread
CUTThread start_thread(CUT_THREADROUTINE func, void * data){
    pthread_t thread;
    pthread_create(&thread, NULL, func, data);
    return thread;
}

//Wait for thread to finish
void end_thread(CUTThread thread){
    pthread_join(thread, NULL);
}

//Destroy thread
void destroy_thread( CUTThread thread ){
    pthread_cancel(thread);
}

//Wait for multiple threads
void wait_for_threads(const CUTThread * threads, int num){
    for(int i = 0; i < num; i++)
        end_thread( threads[i] );
}

void* init_msm_params(void* params)
{

}

struct Mem
{
    size_t device_id;
    void* mem;
};

__global__ void test_one()
{

    Point g, s, t;
    g = g.one();
    printf("One\n");
    g.print();
    printf("Well formed: %d\n", g.is_well_formed());

    s = g.dbl();
    printf("Two\n");
    s.to_affine();
    s.print();
    printf("Well formed: %d\n", s.is_well_formed());
    s = s + g;
    printf("Three\n");
    s.to_affine();
    s.print();

}

int main(int argc, char* argv[])
{
    printf("Hello world\n");
   
    test_one<<<1,1>>>();
    hipDeviceSynchronize();
    hipError_t t = hipGetLastError();
    if(t != hipSuccess)
    {
        printf("Cuda Error: %s \n", hipGetErrorString(t));
        printf("Peek: %s\n", hipGetErrorString(hipPeekAtLastError()));
    }
    return 0;
}

#endif