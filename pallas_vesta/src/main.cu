#include "hip/hip_runtime.h"
#ifndef __MAIN_RUN
#define __MAIN_RUN

#include <stdio.h>
#include <iostream>

#include "./../include/Field.cuh"
#include "./../include/Point.cuh"

#include "./msm.cu"

#include "./../constants/msm_sage_values_2.cuh"

#define debug 1

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " \
                      << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#include <pthread.h>

typedef pthread_t CUTThread;
typedef void *(*CUT_THREADROUTINE)(void *);

#define CUT_THREADPROC void
#define  CUT_THREADEND

//Create thread
CUTThread start_thread(CUT_THREADROUTINE func, void * data){
    pthread_t thread;
    pthread_create(&thread, NULL, func, data);
    return thread;
}

//Wait for thread to finish
void end_thread(CUTThread thread){
    pthread_join(thread, NULL);
}

//Destroy thread
void destroy_thread( CUTThread thread ){
    pthread_cancel(thread);
}

//Wait for multiple threads
void wait_for_threads(const CUTThread * threads, int num){
    for(int i = 0; i < num; i++)
        end_thread( threads[i] );
}

void* init_msm_params(void* params)
{

}

struct Mem
{
    size_t device_id;
    void* mem;
};

__global__ void test_one()
{

    Point g, s, t;
    g = g.one();
    printf("One\n");
    g.print();
    printf("Well formed: %d\n", g.is_well_formed());

    s = g.dbl();
    printf("Two\n");
    s.to_affine();
    s.print();
    printf("Well formed: %d\n", s.is_well_formed());
    s = s + g;
    printf("Three\n");
    s.to_affine();
    s.print();

}
__global__ void init_points_scalars(Scalar *scalar, Point *point, size_t num, uint64_t *sage_scalars, uint64_t *sage_points)
{
    size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    while(idx < num)
    {
        scalar[idx] = Scalar(&sage_scalars[idx*4], 4);
        point[idx].X = Field(&sage_points[2*4*idx]);
        point[idx].Y = Field(&sage_points[2*4*idx + 4]);
        point[idx].Z = point[idx].Z.zero();
        point[idx].to_affine();
        idx += stride;
    }
}

#if debug

__global__ void check_construction(Point *point, Scalar *scalar)
{
    for(int i=0; i< 10; i++)
    {
        point[i].print();
        scalar[i].print();
    }
}

#endif

__global__ void init_points_from_sage(Point *p, size_t num)
{
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    while(idx < num) {
        p[idx].X.encode_montgomery();
        p[idx].Y.encode_montgomery();
        p[idx].Z.encode_montgomery();

        idx += stride;
    }
}

__global__ void init_sage_result(Point *p, const uint64_t *x, const uint64_t *y)
{
    p->X = Field(x);
    p->Y = Field(y);
    p->Z = p->Z.one();
    p->print();
}

int main(int argc, char* argv[])
{

    if (argc < 2) {
		printf("Please enter the MSM scales (e.g. 20 represents 2^20) \n");
		return 1;
	}

    int log_size = atoi(argv[1]);

    size_t num_v = (size_t) (1 << log_size);

    Point *points;
    Scalar *scalars;
    CUDA_CHECK(hipMalloc(&points, sizeof(Point)*num_v));
    CUDA_CHECK(hipMalloc(&scalars, sizeof(Scalar)*num_v));

    CUDA_CHECK(hipMemcpy(points, sage_points, sizeof(uint64_t)*num_v*4*3, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(scalars, sage_scalars, sizeof(uint64_t)*num_v*4, hipMemcpyHostToDevice));

    init_points_from_sage<<<512,128>>>(points, num_v);
    CUDA_CHECK(hipDeviceSynchronize());

    // check_construction<<<1,1>>>(points, scalars);

    cuda_pippenger_msm(points, scalars, num_v);
    CUDA_CHECK(hipDeviceSynchronize());

    Point *sage_res;
    CUDA_CHECK(hipMalloc(&sage_res, sizeof(Point)));
    // init_sage_result<<<1,1>>>(sage_res, sage_msm_result[0], sage_msm_result[1]);
    // CUDA_CHECK(hipDeviceSynchronize());
    printf("Sage Result:\n");
    for(int i=0; i< 2; i++)
    {
        if(!i) printf("X = \n");
        else printf("Y = \n");
        for(int j=3; j>=0 ; j--)
            printf("%016lx ", sage_msm_result[i][j]);
        printf("\n");
    }
    hipError_t t = hipGetLastError();
    if(t != hipSuccess)
    {
        printf("Cuda Error: %s \n", hipGetErrorString(t));
        printf("Peek: %s\n", hipGetErrorString(hipPeekAtLastError()));
    }
    return 0;
}

#endif