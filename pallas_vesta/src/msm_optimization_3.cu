#include "hip/hip_runtime.h"
#ifndef __MSM_CUH
#define __MSM_CUH

#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cuda/pipeline>

#include <hipcub/hipcub.hpp>

#include "./../include/Point.cuh"
#include <vector>

#define debug 1

#define WINDOW_SIZE 16
#define NUM_BITS 256
#define CUDA_CHECK(call)                                                         \
    do                                                                           \
    {                                                                            \
        hipError_t err = call;                                                  \
        if (err != hipSuccess)                                                  \
        {                                                                        \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " \
                      << hipGetErrorString(err) << std::endl;                   \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    } while (0)

__global__ void process_scalar_into_bucket(const Scalar *scalar,
                                           Point *points,
                                           size_t num_points,
                                           size_t num_window,
                                           uint32_t *scalar_chunks,
                                           uint32_t *indices,
                                           uint32_t *offset,
                                           uint32_t *count)
{
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    while (idx < num_points)
    {

        for (int current_window = 0; current_window < num_window; current_window++)
        {
            size_t bindex = 0;
            size_t start = current_window * WINDOW_SIZE;
            size_t end = start + WINDOW_SIZE;
            for (size_t i = start, j = 0; i < end; i++, j++)
            {
                if (scalar[idx].test_bit(i))
                {
                    bindex |= (1 << j);
                }
            }
            scalar_chunks[idx + current_window * num_points] = bindex;
            if (bindex != 0)
                atomicAdd(&count[bindex + current_window * ((size_t)1 << WINDOW_SIZE)], 1);
        }

        idx += stride;
    }
}

__global__ void construct_bucket_indices(
    const __restrict__ uint32_t *scalar_chunks,
    uint32_t *indices,
    uint32_t *offset_counter,
    size_t num_points,
    size_t num_bucket)
{
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    size_t curr_window = blockIdx.y;
    while (idx < num_points)
    {
        uint32_t bindex = scalar_chunks[idx + curr_window * num_points];
        if (bindex != 0)
            indices[atomicAdd(&offset_counter[bindex + curr_window * num_bucket], 1)] = idx;
        idx += stride;
    }
}

__global__ void sum_small_bucket(const Point *point, Point *sum, const uint32_t *offset, const uint32_t *indices,
                                 const uint32_t *count, size_t num_bucket)
{
    extern __shared__ uint32_t tile_idx[];
    size_t bucket = blockIdx.x * blockDim.x + threadIdx.x;
    size_t curr_window = blockIdx.y;
    size_t stride = gridDim.x * blockDim.x;
    
    while (bucket < num_bucket) {
        size_t curr_bucket = bucket + curr_window * num_bucket;
        uint32_t n = count[curr_bucket];
        if (n == 0 || n >= 128) return;
    
        // load indices into shared mem tile
        for (uint32_t i = threadIdx.x; i < n; i += blockDim.x) {
            tile_idx[i] = indices[offset[curr_bucket] + i];
        }
        __syncthreads();
    
        Point acc; acc = acc.zero();
        for (uint32_t i = 0; i < n; ++i) {
            Point p = __ldg(&point[tile_idx[i]]);
            acc = acc.mixed_add(p);
        }
        sum[curr_bucket] = acc;

        bucket += stride;
    } 
    
}
__global__ void sum_medium_bucket(const Point *point, Point *sum, const uint32_t *offset,
                                  const uint32_t *indices, const uint32_t *count, size_t num_bucket)
{
    size_t bucket = blockIdx.x;
    size_t curr_window = blockIdx.y;
    size_t stride = gridDim.x;
    size_t num_threads = blockDim.x;

    extern __shared__ Point shared_sum[];

    while (bucket < num_bucket)
    {
        size_t curr_count = count[bucket + curr_window * num_bucket];
        if (curr_count < 128 || curr_count > 256 * 128)
        {
            bucket += stride;
            continue;
        }
        size_t per_thread = (curr_count + num_threads - 1) / num_threads;

        size_t idx = threadIdx.x;

        size_t start = offset[bucket + curr_window * num_bucket] + idx * per_thread;
        size_t end = start + per_thread;
        if (end > offset[bucket + curr_window * num_bucket] + curr_count)
            end = offset[bucket + curr_window * num_bucket] + curr_count;
        Point lsum;
        lsum = lsum.zero();
        for (size_t i = start; i < end; i++)
        {
            cuda::pipeline<cuda::thread_scope_thread> pipe = cuda::make_pipeline();
            Point *smem_ptr = shared_sum + idx;
            pipe.producer_acquire();
            cuda::memcpy_async(smem_ptr, &point[indices[i]], sizeof(Point), pipe);
            pipe.producer_commit();

            pipe.consumer_wait();
            Point val = *smem_ptr;
            pipe.consumer_release();
            
            lsum = lsum.mixed_add(val);
        }
        shared_sum[idx] = lsum;
        __syncthreads();

        for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
        {
            if (threadIdx.x < stride)
                shared_sum[threadIdx.x] = shared_sum[threadIdx.x] + shared_sum[threadIdx.x + stride];
            __syncthreads();
        }
        if (idx == 0)
        {
            sum[bucket + curr_window * num_bucket] = shared_sum[0];
        }

        bucket += stride;
    }
}


__global__ void gather_bucket_parallel(Point *sum, size_t bcount, Point *res)
{
    int idx = threadIdx.x;
    int curr_window = blockIdx.x;
    extern __shared__ Point local_sum[];
    Point running_sum;
    Point result;
    int per_thread = (bcount + blockDim.x - 1) / blockDim.x;
    int start = idx * per_thread + 1;
    int end = start + per_thread;

    if (end > bcount)
        end = bcount;
    local_sum[idx] = local_sum[idx].zero();
    running_sum = running_sum.zero();
    result = result.zero();

    for (int i = end - 1; i >= start; i--)
    {
        running_sum = running_sum + sum[i + curr_window * bcount];
        local_sum[idx] = local_sum[idx] + running_sum;
    }

    __syncthreads();

    local_sum[idx] = local_sum[idx] + running_sum * (per_thread * idx);
    __syncthreads();
    if (idx % 2 == 0)
    {
        local_sum[idx] = local_sum[idx] + local_sum[idx + 1];
    }
    __syncthreads();
    if (idx % 4 == 0)
    {
        local_sum[idx] = local_sum[idx] + local_sum[idx + 2];
    }
    __syncthreads();
    if (idx % 8 == 0)
    {
        local_sum[idx] = local_sum[idx] + local_sum[idx + 4];
    }
    __syncthreads();
    if (idx % 16 == 0)
    {
        local_sum[idx] = local_sum[idx] + local_sum[idx + 8];
    }
    __syncthreads();
    if (idx % 32 == 0)
    {
        local_sum[idx] = local_sum[idx] + local_sum[idx + 16];
    }
    __syncthreads();
    if (idx == 0)
    {
        for (int i = 0; i < blockDim.x; i += 32)
        {
            result = result + local_sum[i];
        }
        res[curr_window] = result;
    }
}
// accumulate all window output
__global__ void accumulate_result(Point *window_res, size_t num_window, Point *res)
{
    Point acc = acc.zero();
    for (int i = num_window - 1; i >= 0; i--)
    {
        for (int j = 0; j < WINDOW_SIZE; j++)
        {
            acc = acc.dbl();
        }
        acc = acc + window_res[i];
    }
    *res = acc;
}

#if debug
__global__ void print_point(Point *p, size_t num = 1)
{
    for (size_t i = 0; i < num; i++)
    {
        if (!p[i].is_zero())
        {
            p[i].to_affine();
            p[i].print();
        }
    }
}

#endif

// driver function to perform multi scalar multiplication
Point* cuda_pippenger_msm(Point *points, Scalar *scalars, size_t num_points)
{
    int num_windows = (NUM_BITS + WINDOW_SIZE - 1) / WINDOW_SIZE;
    size_t num_bucket = ((size_t)1 << WINDOW_SIZE);

    uint32_t *scalar_chunks, *indices; // scalar_chunks put all the scalar chunks in a single array
    // indices array to store the indices of scalars according to bucket
    uint32_t *offset, *offset_counter; // offset for bucket
    uint32_t *count;                   // count for bucket
    uint32_t *h_count;

    hipStream_t memcpy_stream;


    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    int sm_count = prop.multiProcessorCount;

    CUDA_CHECK(hipStreamCreateWithFlags(&memcpy_stream, hipStreamNonBlocking));
    
    CUDA_CHECK(hipMalloc(&scalar_chunks, sizeof(uint32_t) * num_points * num_windows));
    CUDA_CHECK(hipMalloc(&indices, sizeof(uint32_t) * num_points * num_windows));
    CUDA_CHECK(hipMalloc(&offset, sizeof(uint32_t) * num_bucket * num_windows));
    CUDA_CHECK(hipMalloc(&offset_counter, sizeof(uint32_t) * num_bucket * num_windows));
    CUDA_CHECK(hipMalloc(&count, sizeof(uint32_t) * num_bucket * num_windows));
    CUDA_CHECK(hipMemset(count, 0, sizeof(uint32_t) * num_bucket * num_windows));
    CUDA_CHECK(hipMemset(offset, 0, sizeof(uint32_t) * num_bucket * num_windows));
    CUDA_CHECK(hipMemset(offset_counter, 0, sizeof(uint32_t) * num_bucket * num_windows));
    CUDA_CHECK(hipMemset(scalar_chunks, 0, sizeof(uint32_t) * num_points * num_windows));
    CUDA_CHECK(hipMemset(indices, 0, sizeof(uint32_t) * num_points * num_windows));

    h_count = new uint32_t[num_bucket * num_windows];

    size_t blockSize = 256;
    size_t gridSize = (num_points + blockSize - 1) / blockSize;

    process_scalar_into_bucket<<<gridSize, blockSize>>>(scalars, points, num_points, num_windows, scalar_chunks, indices, offset, count);
    CUDA_CHECK(hipDeviceSynchronize());

    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;

    hipcub::DeviceScan::ExclusiveSum(
        d_temp_storage, temp_storage_bytes, count, offset, num_bucket * num_windows);
    CUDA_CHECK(hipMalloc(&d_temp_storage, temp_storage_bytes));
    hipcub::DeviceScan::ExclusiveSum(
        d_temp_storage, temp_storage_bytes, count, offset, num_bucket * num_windows);
    CUDA_CHECK(hipFree(d_temp_storage));

    CUDA_CHECK(hipMemcpy(offset_counter, offset, sizeof(uint32_t) * num_bucket * num_windows, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipDeviceSynchronize());
    dim3 grid_size(gridSize, num_windows);
    construct_bucket_indices<<<grid_size, blockSize, 0, 0>>>(scalar_chunks, indices, offset_counter, num_points, num_bucket);
    CUDA_CHECK(hipDeviceSynchronize());
    
    Point *sum;
    CUDA_CHECK(hipMalloc(&sum, sizeof(Point) * num_bucket * num_windows));
    dim3 block(128);
    dim3 grid(sm_count*2, num_windows); //(num_bucket + blockSize - 1) / blockSize
    sum_small_bucket<<<grid, block, block.x * sizeof(Point)>>>(points, sum, offset, indices, count, num_bucket);
    sum_medium_bucket<<<grid, block, block.x * sizeof(Point)>>>(points, sum, offset, indices, count, num_bucket);
    CUDA_CHECK(hipDeviceSynchronize());

    Point *window_res;
    CUDA_CHECK(hipMalloc(&window_res, sizeof(Point) * num_windows));
    dim3 gather_grid(num_windows, 1);
    CUDA_CHECK(hipDeviceSynchronize());
    gather_bucket_parallel<<<gather_grid, 256, 48 * 1024>>>(sum, num_bucket, window_res);
    CUDA_CHECK(hipDeviceSynchronize());

    Point *res;
    CUDA_CHECK(hipMalloc(&res, sizeof(Point)));
    accumulate_result<<<1, 1>>>(window_res, num_windows, res);
    CUDA_CHECK(hipDeviceSynchronize());

    // Free memory
    CUDA_CHECK(hipFree(scalar_chunks));
    CUDA_CHECK(hipFree(indices));
    CUDA_CHECK(hipFree(offset));
    CUDA_CHECK(hipFree(offset_counter));
    CUDA_CHECK(hipFree(count));
    CUDA_CHECK(hipFree(sum));
    CUDA_CHECK(hipFree(window_res));

    return res;
}

#endif