#ifndef __FIELD_CU
#define __FIELD_CU

#include "./../include/Field.cuh"

#include "./../utils/field-helper.cuh"

// Constructor without any argument
__host__ __device__ Field::Field()
{
    this->data[0] = 0;
    this->data[1] = 0;
    this->data[2] = 0;
    this->data[3] = 0;
}

__host__ __device__ Field::Field(u_int64_t *uint64_le, size_t len)
{
    copy_limbs(data, uint64_le, len);
    encode_montgomery();
}

__host__ __device__ Field::~Field()
{
#ifdef __CUDA_ARCH__
    free(data);
#else
    hipFree(data);
#endif
}

// Relational Operators
__device__ bool Field::operator==(const Field &other)
{
    return equal(data, other.data, LIMBS);
}
__device__ bool Field::operator==(const Field &other) const
{
    return equal(data, other.data, LIMBS);
}
__device__ bool Field::operator!=(const Field &other)
{
    return !(operator==(other));
}
__device__ bool Field::operator!=(const Field &other) const
{
    return !(operator==(other));
}

__device__ bool is_greater_than_or_equal(const u_int64_t *a, const u_int64_t *b)
{
    for (int i = LIMBS - 1; i >= 0; i--)
    {
        if (a[i] > b[i])
            return true;
        if (a[i] < b[i])
            return false;
    }
    return true;
}
__device__ bool Field::operator>=(const Field &other)
{
    return is_greater_than_or_equal(data, other.data);
}
__device__ bool Field::operator>=(const Field &other) const
{
    return is_greater_than_or_equal(data, other.data);
}

__device__ bool is_less_than_or_equal(const u_int64_t *a, const u_int64_t *b)
{
    for (int i = LIMBS - 1; i >= 0; i--)
    {
        if (a[i] < b[i])
            return true;
        if (a[i] > b[i])
            return false;
    }
    return true;
}

__device__ bool Field::operator<=(const Field &other)
{
    return is_less_than_or_equal(data, other.data);
}
__device__ bool Field::operator<=(const Field &other) const
{
    return is_less_than_or_equal(data, other.data);
}

// Arithmatic operators
__device__ Field &Field::operator+=(const Field &other)
{
    add_limbs(this->data, this->data, other.data, LIMBS);
    conditional_subtract(this->data, pallas::MODULUS, LIMBS);
    return *this;
}
__device__ Field &Field::operator-=(const Field &other)
{
    __uint64_t res[LIMBS];
    bool borrow = sub_limbs(res, this->data, other.data, LIMBS);
    if (borrow)
    {
        add_limbs(this->data, pallas::MODULUS, this->data, LIMBS);
        sub_limbs(this->data, this->data, other.data, LIMBS);
    }
    else
    {
        copy_limbs(this->data, res, LIMBS);
    }
    conditional_subtract(this->data, pallas::MODULUS, LIMBS);
    return *this;
}
__device__ Field &Field::operator*=(const Field &other)
{
    __uint64_t res[2 * LIMBS];
    mont_mul(res, this->data, other.data, pallas::MODULUS, pallas::INV, LIMBS);
    conditional_subtract(res, pallas::MODULUS, LIMBS);
    copy_limbs(data, res, LIMBS);
    return *this;
}
__device__ Field Field::operator+(const Field &other)
{
    Field result;
    add_limbs(result.data, this->data, other.data, LIMBS);
    conditional_subtract(result.data, pallas::MODULUS, LIMBS);
    return result;
}
__device__ Field Field::operator+(const Field &other) const
{
    Field result;
    add_limbs(result.data, this->data, other.data, LIMBS);
    conditional_subtract(result.data, pallas::MODULUS, LIMBS);
    return result;
}
__device__ Field Field::operator-(const Field &other)
{
    Field result(*this);
    result -= other;
    return result;
}
__device__ Field Field::operator-(const Field &other) const
{
    Field result(*this);
    result -= other;
    return result;
}
__device__ Field Field::operator*(const Field &other)
{
    Field result(*this);
    result *= other;
    return result;
}
__device__ Field Field::operator*(const Field &other) const
{
    Field result(*this);
    result *= other;
    return result;
}

// Negation operator
__device__ Field Field::operator-()
{
    if (this->is_zero())
        return *this;

    Field result;
    sub_limbs(result.data, pallas::MODULUS, this->data, LIMBS);
    return result;
}
__device__ Field Field::operator-() const
{
    if (this->is_zero())
        return *this;

    Field result;
    sub_limbs(result.data, pallas::MODULUS, this->data, LIMBS);
    return result;
}

// double the field element
__device__ Field Field::dbl()
{
    Field result;
    for (size_t i = LIMBS - 1; i >= 1; i--)
        result.data[i] = (this->data[i] << 1) | (this->data[i - 1] >> (64 - 1));

    result.data[0] = this->data[0] << 1;
    conditional_subtract(result.data, pallas::MODULUS, LIMBS);
    return result;
}
// square of the field element
__device__ Field Field::squared()
{
    return (*this) * (*this);
}
__device__ Field Field::squared() const
{
    return (*this) * (*this);
}
// clear the set Field values
__device__ void Field::clear()
{
    for (int i = 0; i < LIMBS; i++)
        data[i] = 0;
}
// Check if equal to zero
__device__ bool Field::is_zero()
{
    return is_zero_limbs(data, LIMBS);
}
__device__ bool Field::is_zero() const
{
    return is_zero_limbs(data, LIMBS);
}

// go to montgomery representation
__device__ inline void Field::encode_montgomery()
{
    mont_encode(this->data, this->data, pallas::R2, pallas::MODULUS, pallas::INV, LIMBS);
}
// get out of montgomery representation
__device__ inline void Field::decode_montgomery()
{
    mont_decode(this->data, this->data, pallas::MODULUS, pallas::INV, LIMBS);
}
// find the inverse of the field element
__device__ Field Field::inverse()
{
}

__host__ __device__ void Field::print()
{
    for (int i = LIMBS - 1; i >= 0; i--)
        printf("%016lx ", data[i]);
    printf("\n");
}

#endif